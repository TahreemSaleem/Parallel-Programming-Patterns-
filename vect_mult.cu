
#include "hip/hip_runtime.h"
#include <time.h>
#include <stdio.h>
#include <math.h>
#define TILE_WIDTH 2
__global__
void reduce(int *data, int *result, int N) {
  
  int i = threadIdx.x ;

  
	for (int k = N/2; k > 0; k=k/2) {

		if(i<k)

		data[i] += data[k+i];
		__syncthreads();
	}
        if (i == 0)
		result[0] = data[0]; 

}
__global__
void map(int *vecA, int *vecB, int *vecC) {
  
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  vecC[i] = vecA[i] * vecB[i] ;
  
}



int main() {

   int N =65536;

  int *A_h = (int*)malloc(sizeof(int) * N );
  int *B_h = (int*)malloc(sizeof(int) * N );
  int *C_h = (int*)malloc(sizeof(int) * N );
  int *D_h = (int*)malloc(sizeof(int) * N );
  for (int i=0; i< N; i++) A_h[i] = i;
  /*for (int i = 0; i < N; i++) {
		
		printf("%d \t", A_h[i]);
	}
	printf("\n");*/
  for (int i=0; i< N; i++) B_h[i] = i;
  /*for (int i = 0; i < N; i++) {
		
		printf("%d \t", B_h[i]);
	}
	printf("\n");*/
  //for (int i=0; i< N; i++) fprintf(stdout, "%f\n", A_h[i]);
  clock_t begin, end;
  double elapsed;

  //initialize matrices

  int *A_d, *B_d,*C_d,*D_d;
  hipMalloc(&A_d, sizeof(int) * N );
  hipMalloc(&B_d, sizeof(int) * N);
  hipMalloc(&C_d, sizeof(int) * N);
  hipMalloc(&D_d, sizeof(int) * N);

  begin = clock();

  hipMemcpy(A_d, A_h, sizeof(int) * N , hipMemcpyHostToDevice);
  hipMemcpy(B_d, B_h, sizeof(int) * N , hipMemcpyHostToDevice);

  
  //launch kernel
  //dim3 dimBlock(2, 2);
  //dim3 dimGrid(N/2, N/2);
   int xBlock = (N/TILE_WIDTH);
   int xGrid = 1; 

//matrixMultSimple<<<dimGrid, dimBlock>>>(A_d, B_d, C_d, N);

  //reduce<<<xGrid, xBlock>>>(A_d, B_d,  N, N);
    map<<<xGrid, xBlock>>>(A_d, B_d, C_d);
hipMemcpy(C_h, C_d, sizeof(float) * N, hipMemcpyDeviceToHost);  
    /* for (int i = 0; i < N; i++) {
		
		printf("%d \t", C_h[i]);
	}
	printf("\n");
*/
  reduce<<<xGrid, xBlock>>>(C_d,D_d ,N);

 
hipMemcpy(D_h, D_d, sizeof(float) * N, hipMemcpyDeviceToHost);  
// fprintf(stdout, "%d\n", D_h[0]);
  end = clock();
  elapsed  = double(end - begin)/CLOCKS_PER_SEC;

  fprintf(stdout, "%f\n", elapsed);
  hipFree(A_d);
  hipFree(B_d);
 
  free(A_h);
  free(B_h);
 

  return 0;
}
