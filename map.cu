
#include "hip/hip_runtime.h"
#include <time.h>
#include <stdio.h>
#include <math.h>

__global__
void map(float *data, float *result, int N) {
  
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  result[i] = pow(data[i],2) ;
  //result[i]= 2;

}




int main() {

  const int N = 10;

  float *A_h = (float*)malloc(sizeof(float) * N *N);
  float *B_h = (float*)malloc(sizeof(float) * N *N);

  
  for (int i=0; i< N; i++) A_h[i] = 5;


  //for (int i=0; i< N; i++) fprintf(stdout, "%f\n", A_h[i]);
  //clock_t begin, end;
  //double elapsed;

  //initialize matrices

  float *A_d, *B_d;
  hipMalloc(&A_d, sizeof(float) * N * N );
  hipMalloc(&B_d, sizeof(float) * N * N );


  //begin = clock();

  hipMemcpy(A_d, A_h, sizeof(float) * N * N , hipMemcpyHostToDevice);


  
  //launch kernel
  dim3 dimBlock(10, 10);
  dim3 dimGrid(N/10, N/10);
  //matrixMultSimple<<<dimGrid, dimBlock>>>(A_d, B_d, C_d, N);
 map<<<dimGrid, dimBlock>>>(A_d, B_d,  N);

  hipMemcpy(B_h, B_d, sizeof(float) * N *N , hipMemcpyDeviceToHost);
  for (int i=0; i< N; i++) fprintf(stdout, "%f\n", B_h[i]);
  //end = clock();
  //elapsed  = double(end - begin)/CLOCKS_PER_SEC;

  //fprintf(stdout, "%f\n", elapsed);
  hipFree(A_d);
  hipFree(B_d);
 
  free(A_h);
  free(B_h);
 

  return 0;
}
