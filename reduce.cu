
#include "hip/hip_runtime.h"
#include <time.h>
#include <stdio.h>
#include <math.h>
#define TILE_WIDTH 2
__global__
void reduce(int *data, int *result, int N) {
  
  int i = threadIdx.x ;

  /*int k ;
  for (k = 0; k <TILE_WIDTH;k++ )
      {  
	__syncthreads();
	result[i] = result[i] + data[i+ k * (N/TILE_WIDTH)];
        __syncthreads();
      }*/

	for (int k = N/2; k > 0; k=k/2) {
		if(i<k)
		data[i] += data[k+i];
		__syncthreads();
	}
        if (i == 0)
		result[0] = data[0]; 

}




int main() {

   int N =8;

  int *A_h = (int*)malloc(sizeof(int) * N );
  int *B_h = (int*)malloc(sizeof(int) * N );

  
  for (int i=0; i< N; i++) A_h[i] = i;
  for (int i = 0; i < N; i++) {
		
		printf("%d \t", A_h[i]);
	}
	printf("\n");

  //for (int i=0; i< N; i++) fprintf(stdout, "%f\n", A_h[i]);
  //clock_t begin, end;
  //double elapsed;

  //initialize matrices

  int *A_d, *B_d;
  hipMalloc(&A_d, sizeof(int) * N );
  hipMalloc(&B_d, sizeof(int) * N);
  


  //begin = clock();

  hipMemcpy(A_d, A_h, sizeof(int) * N , hipMemcpyHostToDevice);


  
  //launch kernel
  //dim3 dimBlock(2, 2);
  //dim3 dimGrid(N/2, N/2);
   int xBlock = (N/TILE_WIDTH);
   int xGrid = 1;  

//matrixMultSimple<<<dimGrid, dimBlock>>>(A_d, B_d, C_d, N);

  //reduce<<<xGrid, xBlock>>>(A_d, B_d,  N, N);
  reduce<<<xGrid, xBlock>>>(A_d, B_d,  N);

 
hipMemcpy(B_h, B_d, sizeof(float) * N, hipMemcpyDeviceToHost);  
 fprintf(stdout, "%d\n", B_h[0]);
  //end = clock();
  //elapsed  = double(end - begin)/CLOCKS_PER_SEC;

 // fprintf(stdout, "%d\n", elapsed);
  hipFree(A_d);
  hipFree(B_d);
 
  free(A_h);
  free(B_h);
 

  return 0;
}
